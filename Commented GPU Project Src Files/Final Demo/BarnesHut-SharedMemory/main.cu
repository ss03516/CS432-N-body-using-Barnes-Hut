#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.cu"

/******************************************************************************/

static void CudaTest(const char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}


/******************************************************************************/

// random number generator

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

static int A = 1;
static int B = 0;
static int randx = 1;
static int lastrand;


static void drndset(int seed)
{
   A = 1;
   B = 0;
   randx = (A * seed + B) & MASK;
   A = (MULT * A) & MASK;
   B = (MULT * B + ADD) & MASK;
}


static double drnd()
{
   lastrand = randx;
   randx = (A * randx + B) & MASK;
   return (double)lastrand / TWOTO31;
}


/******************************************************************************/

int main(int argc, char *argv[])
{
  register int i, run, blocks;
  int nnodes, nbodies, step, timesteps;
  register double runtime;
  register float dtime, dthf, epssq, itolsq;
  float time, timing[7];
  hipEvent_t start, stop;
  float *mass, *posx, *posy, *posz, *velx, *vely, *velz;

  int  *sortl, *childl, *countl, *startl;
  float *massl;
  float *posxl, *posyl, *poszl;
  float *velxl, *velyl, *velzl;
  float *accxl, *accyl, *acczl;
  float *maxxl, *maxyl, *maxzl;
  float *minxl, *minyl, *minzl;
  register double rsc, vsc, r, v, x, y, z, sq, scale;

  // perform some checks

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (argc != 4) {
    fprintf(stderr, "\n");
    fprintf(stderr, "arguments: number_of_bodies number_of_timesteps device\n");
    exit(-1);
  }
  
  
  printf("Device count: %d",deviceCount);  
  const int dev = atoi(argv[3]);
  if ((dev < 0) || (deviceCount <= dev)) {
    fprintf(stderr, "There is no device %d\n", dev);
    exit(-1);
  }
  hipSetDevice(dev);

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  
  
  

  blocks = deviceProp.multiProcessorCount;
//  fprintf(stderr, "blocks = %d\n", blocks);

  
  if (MAXDEPTH > WARPSIZE) {
    fprintf(stderr, "MAXDEPTH must be less than or equal to WARPSIZE\n");
    exit(-1);
  }
  if ((T1 <= 0) || (T1 & (T1-1) != 0)) {
    fprintf(stderr, "T1 must be greater than zero and a power of two\n");
    exit(-1);
  }

  // set L1/shared memory configuration
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(InitializationKernel1), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(InitializationKernel2), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CoGKernel), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(UpdateKernel), hipFuncCachePreferL1);
  hipGetLastError();  // reset error value
  for (run = 0; run < 1; run++) {
    for (i = 0; i < 7; i++) timing[i] = 0.0f;

    nbodies = atoi(argv[1]);
    if (nbodies < 1) {
      fprintf(stderr, "nbodies is too small: %d\n", nbodies);
      exit(-1);
    }
    if (nbodies > (1 << 30)) {
      fprintf(stderr, "nbodies is too large: %d\n", nbodies);
      exit(-1);
    }
    nnodes = nbodies * 2;
    if (nnodes < 1024*blocks) nnodes = 1024*blocks;
    while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
    nnodes--;

    timesteps = atoi(argv[2]);
    dtime = 0.025;  dthf = dtime * 0.5f;
    epssq = 0.05 * 0.05;
    itolsq = 1.0f / (0.5 * 0.5);

    // allocate memory

    if (run == 0) {
      printf("configuration: %d bodies, %d time steps\n", nbodies, timesteps);

      mass = (float *)malloc(sizeof(float) * nbodies);
      if (mass == NULL) {fprintf(stderr, "cannot allocate mass\n");  exit(-1);}
      posx = (float *)malloc(sizeof(float) * nbodies);
      if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
      posy = (float *)malloc(sizeof(float) * nbodies);
      if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
      posz = (float *)malloc(sizeof(float) * nbodies);
      if (posz == NULL) {fprintf(stderr, "cannot allocate posz\n");  exit(-1);}
      velx = (float *)malloc(sizeof(float) * nbodies);
      if (velx == NULL) {fprintf(stderr, "cannot allocate velx\n");  exit(-1);}
      vely = (float *)malloc(sizeof(float) * nbodies);
      if (vely == NULL) {fprintf(stderr, "cannot allocate vely\n");  exit(-1);}
      velz = (float *)malloc(sizeof(float) * nbodies);
      if (velz == NULL) {fprintf(stderr, "cannot allocate velz\n");  exit(-1);}

      //if (hipSuccess != hipMalloc((void **)&errl, sizeof(int))) fprintf(stderr, "could not allocate errd\n");  CudaTest("couldn't allocate errd");
      if (hipSuccess != hipMalloc((void **)&childl, sizeof(int) * (nnodes+1) * 8)) fprintf(stderr, "could not allocate childd\n");  CudaTest("couldn't allocate childd");
      if (hipSuccess != hipMalloc((void **)&massl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate massd\n");  CudaTest("couldn't allocate massd");
      if (hipSuccess != hipMalloc((void **)&posxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posxd\n");  CudaTest("couldn't allocate posxd");
      if (hipSuccess != hipMalloc((void **)&posyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate posyd\n");  CudaTest("couldn't allocate posyd");
      if (hipSuccess != hipMalloc((void **)&poszl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate poszd\n");  CudaTest("couldn't allocate poszd");
      if (hipSuccess != hipMalloc((void **)&velxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate velxd\n");  CudaTest("couldn't allocate velxd");
      if (hipSuccess != hipMalloc((void **)&velyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate velyd\n");  CudaTest("couldn't allocate velyd");
      if (hipSuccess != hipMalloc((void **)&velzl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate velzd\n");  CudaTest("couldn't allocate velzd");
      if (hipSuccess != hipMalloc((void **)&accxl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate accxd\n");  CudaTest("couldn't allocate accxd");
      if (hipSuccess != hipMalloc((void **)&accyl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate accyd\n");  CudaTest("couldn't allocate accyd");
      if (hipSuccess != hipMalloc((void **)&acczl, sizeof(float) * (nnodes+1))) fprintf(stderr, "could not allocate acczd\n");  CudaTest("couldn't allocate acczd");
      if (hipSuccess != hipMalloc((void **)&countl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate countd\n");  CudaTest("couldn't allocate countd");
      if (hipSuccess != hipMalloc((void **)&startl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate startd\n");  CudaTest("couldn't allocate startd");
      if (hipSuccess != hipMalloc((void **)&sortl, sizeof(int) * (nnodes+1))) fprintf(stderr, "could not allocate sortd\n");  CudaTest("couldn't allocate sortd");

      if (hipSuccess != hipMalloc((void **)&maxxl, sizeof(float) * blocks * F1)) fprintf(stderr, "could not allocate maxxd\n");  CudaTest("couldn't allocate maxxd");
      if (hipSuccess != hipMalloc((void **)&maxyl, sizeof(float) * blocks * F1)) fprintf(stderr, "could not allocate maxyd\n");  CudaTest("couldn't allocate maxyd");
      if (hipSuccess != hipMalloc((void **)&maxzl, sizeof(float) * blocks * F1)) fprintf(stderr, "could not allocate maxzd\n");  CudaTest("couldn't allocate maxzd");
      if (hipSuccess != hipMalloc((void **)&minxl, sizeof(float) * blocks * F1)) fprintf(stderr, "could not allocate minxd\n");  CudaTest("couldn't allocate minxd");
      if (hipSuccess != hipMalloc((void **)&minyl, sizeof(float) * blocks * F1)) fprintf(stderr, "could not allocate minyd\n");  CudaTest("couldn't allocate minyd");
      if (hipSuccess != hipMalloc((void **)&minzl, sizeof(float) * blocks * F1)) fprintf(stderr, "could not allocate minzd\n");  CudaTest("couldn't allocate minzd");
    }

    // generate input

    drndset(7);
    rsc = (3 * 3.1415926535897932384626433832795) / 16;
    vsc = sqrt(1.0 / rsc);
    for (i = 0; i < nbodies; i++) {
      mass[i] = 1.0 / nbodies;
      r = 1.0 / sqrt(pow(drnd()*0.999, -2.0/3.0) - 1);
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = rsc * r / sqrt(sq);
      posx[i] = x * scale;
      posy[i] = y * scale;
      posz[i] = z * scale;

      do {
        x = drnd();
        y = drnd() * 0.1;
      } while (y > x*x * pow(1 - x*x, 3.5));
      v = x * sqrt(2.0 / sqrt(1 + r*r));
      do {
        x = drnd()*2.0 - 1.0;
        y = drnd()*2.0 - 1.0;
        z = drnd()*2.0 - 1.0;
        sq = x*x + y*y + z*z;
      } while (sq > 1.0);
      scale = vsc * v / sqrt(sq);
      velx[i] = x * scale;
      vely[i] = y * scale;
      velz[i] = z * scale;
	  // Array containing positions and vellocities in all three dimensions of all the bodies. E.g velx contains all velocities in the x-direction of all the nbodies.
    }

    if (hipSuccess != hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of mass to device failed\n");  CudaTest("mass copy to device failed");
    if (hipSuccess != hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posx to device failed\n");  CudaTest("posx copy to device failed");
    if (hipSuccess != hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posy to device failed\n");  CudaTest("posy copy to device failed");
    if (hipSuccess != hipMemcpy(poszl, posz, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of posz to device failed\n");  CudaTest("posz copy to device failed");
    if (hipSuccess != hipMemcpy(velxl, velx, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velx to device failed\n");  CudaTest("velx copy to device failed");
    if (hipSuccess != hipMemcpy(velyl, vely, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of vely to device failed\n");  CudaTest("vely copy to device failed");
    if (hipSuccess != hipMemcpy(velzl, velz, sizeof(float) * nbodies, hipMemcpyHostToDevice)) fprintf(stderr, "copying of velz to device failed\n");  CudaTest("velz copy to device failed");

    // run timesteps (launch GPU kernels)

    hipEventCreate(&start);  hipEventCreate(&stop);  
    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);

    hipEventRecord(start, 0);
    InitializationKernel<<<1, 1>>>();
    hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    timing[0] += time;
    CudaTest("kernel 0 launch failed");
	printf("Start of timestep \n");
	
    for (step = 0; step < timesteps; step++)
	{
	
	printf("TIMESTEP = %d \n", step);
      hipEventRecord(start, 0);
      BoundingBoxKernel<<<blocks * F1, T1>>>(nnodes, nbodies, startl, childl, massl, posxl, posyl, poszl, maxxl, maxyl, maxzl, minxl, minyl, minzl);
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[1] += time;
      CudaTest("kernel 1 launch failed");

      hipEventRecord(start, 0);
      InitializationKernel1<<<blocks * 1, 1024>>>(nnodes, nbodies, childl);
      TreeBuildingKernel<<<blocks * F2, T2>>>(nnodes, nbodies, childl, posxl, posyl, poszl);
      InitializationKernel2<<<blocks * 1, 1024>>>(nnodes, startl, massl);
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[2] += time;
      CudaTest("kernel 2 launch failed");

      hipEventRecord(start, 0);
      CoGKernel<<<blocks * F3, T3>>>(nnodes, nbodies, countl, childl, massl, posxl, posyl, poszl);
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[3] += time;
      CudaTest("kernel 3 launch failed");

      hipEventRecord(start, 0);
      SortKernel<<<blocks * F4, T4>>>(nnodes, nbodies, sortl, countl, startl, childl);
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[4] += time;
      CudaTest("kernel 4 launch failed");

      hipEventRecord(start, 0);
      ForceKernel<<<blocks * F5, T5>>>(nnodes, nbodies, dthf, itolsq, epssq, sortl, childl, massl, posxl, posyl, poszl, velxl, velyl, velzl, accxl, accyl, acczl);
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[5] += time;
      CudaTest("kernel 5 launch failed");

      hipEventRecord(start, 0);
      UpdateKernel<<<blocks * F6, T6>>>(nbodies, dtime, dthf, posxl, posyl, poszl, velxl, velyl, velzl, accxl, accyl, acczl);
      hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      timing[6] += time;
      CudaTest("kernel 6 launch failed");
    }
    CudaTest("kernel launch failed");
    hipEventDestroy(start);  hipEventDestroy(stop);

    // transfer result back to CPU
    //if (hipSuccess != hipMemcpy(&error, errl, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of err from device failed\n");  CudaTest("err copy from device failed");
    if (hipSuccess != hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posx from device failed\n");  CudaTest("posx copy from device failed");
    if (hipSuccess != hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posy from device failed\n");  CudaTest("posy copy from device failed");
    if (hipSuccess != hipMemcpy(posz, poszl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of posz from device failed\n");  CudaTest("posz copy from device failed");
    if (hipSuccess != hipMemcpy(velx, velxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velx from device failed\n");  CudaTest("velx copy from device failed");
    if (hipSuccess != hipMemcpy(vely, velyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of vely from device failed\n");  CudaTest("vely copy from device failed");
    if (hipSuccess != hipMemcpy(velz, velzl, sizeof(float) * nbodies, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of velz from device failed\n");  CudaTest("velz copy from device failed");

    gettimeofday(&endtime, NULL);
    runtime = endtime.tv_sec + endtime.tv_usec/1000000.0 - starttime.tv_sec - starttime.tv_usec/1000000.0;

    printf("runtime: %.4lf s  (", runtime);
    time = 0;
    for (i = 1; i < 7; i++) {
      printf(" %.1f ", timing[i]);
      time += timing[i];
    }
     printf(") = %.1f ms\n", time);
  }

  // print output
  i = 0;
//  for (i = 0; i < nbodies; i++) {
//    printf("end position of body 0 %.2e %.2e %.2e\n", posx[i], posy[i], posz[i]);
//  }

  free(mass);
  free(posx);
  free(posy);
  free(posz);
  free(velx);
  free(vely);
  free(velz);

  //hipFree(errl);
  hipFree(childl);
  hipFree(massl);
  hipFree(posxl);
  hipFree(posyl);
  hipFree(poszl);
  hipFree(countl);
  hipFree(startl);

  hipFree(maxxl);
  hipFree(maxyl);
  hipFree(maxzl);
  hipFree(minxl);
  hipFree(minyl);
  hipFree(minzl);

  return 0;
}
