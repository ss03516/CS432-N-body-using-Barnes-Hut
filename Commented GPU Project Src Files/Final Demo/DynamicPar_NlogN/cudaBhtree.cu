#include "hip/hip_runtime.h"
#include "cudaOctant.cu"
#include <random>

class Bhtree
{
public:
	body *myBod;
	body *toDelete;
	Octant *octy;
	Bhtree *UNW;
	Bhtree *UNE;
	Bhtree *USW;
	Bhtree *USE;
	Bhtree *DNW;
	Bhtree *DNE;
	Bhtree *DSW;
	Bhtree *DSE;
	int a;

	__host__ __device__ Bhtree(Octant* o)
	{
		myBod = NULL;
		toDelete = NULL;
		octy = o;
		UNW = NULL;
		UNE = NULL;
		USW = NULL;
		USE = NULL;
		DNW = NULL;
		DNE = NULL;
		DSW = NULL;
		DSE = NULL;
		a=10;
	}

	__host__ __device__ ~Bhtree()
	{
		// check if each is ==0 (null)
		if (UNW!=NULL) delete UNW; //UNW->~Bhtree();
		if (UNE!=NULL) delete UNE; //UNE->~Bhtree();
		if (USW!=NULL) delete USW; //USW->~Bhtree();
		if (USE!=NULL) delete USE; //USE->~Bhtree();
		if (DNW!=NULL) delete DNW; //DNW->~Bhtree();
		if (DNE!=NULL) delete DNE; //DNE->~Bhtree();
		if (DSW!=NULL) delete DSW; //DSW->~Bhtree();
		if (DSE!=NULL) delete DSE; //DSE->~Bhtree();
		if (toDelete!=NULL) delete toDelete;
		delete octy; //octy->~Octant();
	//	if (!isExternal() && myBod!= NULL && myBod->velocity.x==0.0) delete myBod;
	//	delete myBod;
	}

	__host__ __device__ bool isExternal()
	{
		return UNW==NULL && UNE==NULL && USW==NULL && USE==NULL &&
			   DNW==NULL && DNE==NULL && DSW==NULL && DSE==NULL;
	}

	__host__ __device__ void insert(body* insertBod)
	{
		if (myBod==NULL)
		{
			myBod = insertBod;
		} else //if (!isExternal())
		{
			bool isExtern = isExternal();
			body *updatedBod;
			if (!isExtern)
			{
				updatedBod = new struct body;
				updatedBod->position.x = (insertBod->position.x*insertBod->mass +
								       myBod->position.x*myBod->mass) /
								  (insertBod->mass+myBod->mass);
				updatedBod->position.y = (insertBod->position.y*insertBod->mass +
									   myBod->position.y*myBod->mass) /
								  (insertBod->mass+myBod->mass);
				updatedBod->position.z = (insertBod->position.z*insertBod->mass +
									   myBod->position.z*myBod->mass) /
								  (insertBod->mass+myBod->mass);
				updatedBod->mass = insertBod->mass+myBod->mass;
			//	delete myBod;
				if (toDelete!=NULL) delete toDelete;
				toDelete = updatedBod;
				myBod = updatedBod;
				updatedBod = insertBod;
			} else {
				updatedBod = myBod;
			}
			Octant *unw = octy->mUNW();
			if (unw->contains(updatedBod->position))
			{
				if (UNW==NULL) { UNW = new Bhtree(unw); }
				else { delete unw; }
				UNW->insert(updatedBod);
			} else {
				delete unw;
				Octant *une = octy->mUNE();
				if (une->contains(updatedBod->position))
				{
					if (UNE==NULL) { UNE = new Bhtree(une); }
					else { delete une; }
					UNE->insert(updatedBod);
				} else {
					delete une;
					Octant *usw = octy->mUSW();
					if (usw->contains(updatedBod->position))
					{
						if (USW==NULL) { USW = new Bhtree(usw); }
						else { delete usw; }
						USW->insert(updatedBod);
					} else {
						delete usw;
						Octant *use = octy->mUSE();
						if (use->contains(updatedBod->position))
						{
							if (USE==NULL) { USE = new Bhtree(use); }
							else { delete use; }
							USE->insert(updatedBod);
						} else {
							delete use;
							Octant *dnw = octy->mDNW();
							if (dnw->contains(updatedBod->position))
							{
								if (DNW==NULL) { DNW = new Bhtree(dnw); }
								else { delete dnw; }
								DNW->insert(updatedBod);
							} else {
								delete dnw;
								Octant *dne = octy->mDNE();
								if (dne->contains(updatedBod->position))
								{
									if (DNE==NULL) { DNE = new Bhtree(dne); }
									else { delete dne; }
									DNE->insert(updatedBod);
								} else {
									delete dne;
									Octant *dsw = octy->mDSW();
									if (dsw->contains(updatedBod->position))
									{
										if (DSW==NULL) { DSW = new Bhtree(dsw); }
										else { delete dsw; }
										DSW->insert(updatedBod);
									} else {
										delete dsw;
										Octant *dse = octy->mDSE();
										if (DSE==NULL) { DSE = new Bhtree(dse); }
										else { delete dse; }
										DSE->insert(updatedBod);
										}
									}
								}
							}
						}
					}
				}
		//	delete updatedBod;
			if (isExtern) {
				insert(insertBod);
			}
		}
	}

	__host__ __device__ double magnitude(vec3* v)
	{
		return sqrt(v->x*v->x+v->y*v->y+v->z*v->z);
	}

	__host__ __device__ double magnitude( double x, double y, double z)
	{
		return sqrt(x*x+y*y+z*z);
	}

	__host__ __device__ void singleInteract(struct body* target, struct body* other)
	{
		vec3 *posDiff = new struct vec3;
		posDiff->x = (target->position.x-other->position.x)*TO_METERS;
		posDiff->y = (target->position.y-other->position.y)*TO_METERS;
		posDiff->z = (target->position.z-other->position.z)*TO_METERS;
		double dist = magnitude(posDiff);
		double F = TIME_STEP*(G*target->mass*other->mass) / ((dist*dist + SOFTENING*SOFTENING) * dist);

		target->accel.x -= F*posDiff->x/target->mass;
		target->accel.y -= F*posDiff->y/target->mass;
		target->accel.z -= F*posDiff->z/target->mass;
		delete posDiff;

	}

	__host__ __device__ void interactInTree(body* bod)
	{
		if (isExternal())
		{
			if (myBod!=bod) { singleInteract(bod, myBod); }
		}
		else if (octy->getLength() /
				magnitude(myBod->position.x-bod->position.x,
						  myBod->position.y-bod->position.y,
						  myBod->position.z-bod->position.z) < MAX_DISTANCE)
		{
			singleInteract(bod, myBod);
		} else {
			if (UNW!=NULL) UNW->interactInTree(bod);
			if (UNE!=NULL) UNE->interactInTree(bod);
			if (USW!=NULL) USW->interactInTree(bod);
			if (USE!=NULL) USE->interactInTree(bod);
			if (DNW!=NULL) DNW->interactInTree(bod);
			if (DNE!=NULL) DNE->interactInTree(bod);
			if (DSW!=NULL) DSW->interactInTree(bod);
			if (DSE!=NULL) DSE->interactInTree(bod);
		}
	}

};


